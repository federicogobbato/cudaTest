#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
int atomicAdd(int* address, int val);
#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "GenericCUDA.h"


__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void cubeKernel(int *d_out, const int *d_in)
{
	int i = threadIdx.x;
	int f = d_in[i];
	d_out[i] = f * f * f;
}

hipError_t cubeWithCuda(int *out, const int *in, unsigned int size) {

	int *d_in = 0;
	int *d_out = 0;
	hipError_t cudaStatus = hipSuccess;

	try
	{
		std::cout << &d_in << std::endl;

		// Choose which GPU to run on, change this on a multi-GPU system.
		checkCudaErrorsAndExit(hipSetDevice(0));

		transferDataHostToDev<int>(in, &d_in, &d_out, size);

		// Launch a kernel on the GPU 
		runKernel<int>(&d_out, &d_in, cubeKernel, size);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw cudaStatus;
		}
	}
	catch (hipError_t ex)
	{
		cudaStatus = ex;
		hipFree(d_in);
		hipFree(d_out);
	}
	hipFree(d_in);
	hipFree(d_out);

	return cudaStatus;
}