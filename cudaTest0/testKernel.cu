#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
int atomicAdd(int* address, int val);
#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "GenericCUDA.h"


__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void cubeKernel(int *d_out, const int *d_in)
{
	int i = threadIdx.x;
	int f = d_in[i];
	d_out[i] = f * f * f;
}

hipError_t cubeWithCuda(int *out, const int *in, unsigned int size) {

	int *d_in = 0;
	int *d_out = 0;
	hipError_t cudaStatus = hipSuccess;

	try
	{
		std::cout << &d_in << std::endl;

		transferDataHostToDev<int>(in, &d_in, &d_out, size);

		// Launch a kernel on the GPU 
		runKernel<int>(&d_out, &d_in, cubeKernel, size);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw cudaStatus;
		}
	}
	catch (hipError_t ex)
	{
		cudaStatus = ex;
		hipFree(d_in);
		hipFree(d_out);
	}
	hipFree(d_in);
	hipFree(d_out);

	return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
////hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
////{
////	int *dev_inA = 0;
////	int *dev_inB = 0;
////	int *dev_outC = 0;
////	hipError_t cudaStatus;
////
////	// Choose which GPU to run on, change this on a multi-GPU system.
////	cudaStatus = hipSetDevice(0);
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
////		goto Error;
////	}
////
////	// Allocate GPU buffers for three vectors (two input, one output)    .
////	cudaStatus = hipMalloc((void**)&dev_outC, size * sizeof(int));
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMalloc failed!");
////		goto Error;
////	}
////
////	cudaStatus = hipMalloc((void**)&dev_inA, size * sizeof(int));
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMalloc failed!");
////		goto Error;
////	}
////
////	cudaStatus = hipMalloc((void**)&dev_inB, size * sizeof(int));
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMalloc failed!");
////		goto Error;
////	}
////
////	// Copy input vectors from host memory to GPU buffers.
////	cudaStatus = hipMemcpy(dev_inA, a, size * sizeof(int), hipMemcpyHostToDevice);
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMemcpy failed!");
////		goto Error;
////	}
////
////	cudaStatus = hipMemcpy(dev_inB, b, size * sizeof(int), hipMemcpyHostToDevice);
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMemcpy failed!");
////		goto Error;
////	}
////
////	// Launch a kernel on the GPU with one thread for each element.
////	addKernel <<<1, size >>> (dev_outC, dev_inA, dev_inB);
////
////	// Check for any errors launching the kernel
////	cudaStatus = hipGetLastError();
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
////		goto Error;
////	}
////
////	// hipDeviceSynchronize waits for the kernel to finish, and returns
////	// any errors encountered during the launch.
////	cudaStatus = hipDeviceSynchronize();
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
////		goto Error;
////	}
////
////	// Copy output vector from GPU buffer to host memory.
////	cudaStatus = hipMemcpy(c, dev_outC, size * sizeof(int), hipMemcpyDeviceToHost);
////	if (cudaStatus != hipSuccess) {
////		fprintf(stderr, "hipMemcpy failed!");
////		goto Error;
////	}
////
////Error:
////	hipFree(dev_outC);
////	hipFree(dev_inA);
////	hipFree(dev_inB);
////
////	return cudaStatus;
////}