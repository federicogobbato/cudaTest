#include "hip/hip_runtime.h"
#pragma once

#ifdef __INTELLISENSE__
void __syncthreads();
int atomicAdd(int* address, int val);
#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "GenericCUDA.h"
#include "utils.h"
#include "timer.h"

const int MAX_THREADS_BLOCK = 256;

template<class T>
__global__ void reduceKernel(T *d_out, const T *d_in, const int size)
{
	extern __shared__ T sdata[];

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	if (id < size) {
		// load shared mem from global mem and, make sure entire block is loaded!
		sdata[tid] = d_in[id];
		__syncthreads();

		atomicAdd(&sdata[0], sdata[tid]);
		__syncthreads();

		// only thread 0 writes result for this block back to global mem
		if (tid == 0)
		{
			int blockResult = sdata[0] - d_in[(id / blockDim.x)*blockDim.x];
			d_out[blockIdx.x] = blockResult;
		}
	}
}

//!Seems to be the faster solution for reduce but WORK ONLY for int?!
template<class T>
__global__ void reduceKernel1(T *d_out, const T *d_in, const int size)
{
	extern __shared__ T sdata[];

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	if (id < size) {
		// load shared mem from global mem and, make sure entire block is loaded!
		sdata[tid] = d_in[id];
		__syncthreads();            

		atomicAdd(&sdata[0], sdata[tid]);
		__syncthreads();

		if (tid == 0)
		{
			int blockResult = sdata[0] - d_in[(id / blockDim.x)*blockDim.x];
			atomicAdd(d_out, blockResult);
		}
	}
}

template<class T>
__global__ void reduceKernel2(T *d_out, const T *d_in, int size)
{
	extern __shared__ T sdata[];

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	if (id < size) {
		sdata[tid] = d_in[id];
		__syncthreads();

		for (unsigned int s = 1; s < blockDim.x; s *= 2) {
			if (tid % (2 * s) == 0) {
				sdata[tid] += sdata[tid + s];
			}
			__syncthreads();
		}

		if (tid == 0)
		{
			d_out[blockIdx.x] = sdata[0];
		}
	}
}

template<class T>
__global__ void reduceKernel3(T *d_out, const T *d_in)
{
	extern __shared__ T sdata[];

	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = d_in[myId];
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

hipError_t reduceWithCuda(unsigned int *out, unsigned int *in, int size) {

	const int ARRAY_BYTES = size * sizeof(unsigned int);
	unsigned int* d_in = 0;
	unsigned int* d_out = 0;
	////unsigned int* d_intermediate = 0;
	hipError_t cudaStatus = hipSuccess;

	GpuTimer* timer = new GpuTimer();
	double delay = 0;
	double kernelTime = 0;

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		checkCudaErrorsAndExit(hipSetDevice(0));

		int maxThreads = size;
		int blocks = 1;

		if (size > MAX_THREADS_BLOCK) {
			maxThreads = MAX_THREADS_BLOCK;
			blocks = std::ceil((float)size / maxThreads);
		}

		// Allocate GPU buffers for 2 vectors (one input, one output)    .
		checkCudaErrors(hipMalloc((void**)&d_in, ARRAY_BYTES));
		////checkCudaErrors(hipMalloc((void**)&d_intermediate, ARRAY_BYTES));
		checkCudaErrors(hipMalloc((void**)&d_out, sizeof(unsigned int)));

		timer->Start();
		// Copy input vectors from host memory to GPU buffers.
		checkCudaErrors(hipMemcpy(d_in, in, ARRAY_BYTES, hipMemcpyHostToDevice));
		timer->Stop();
		delay = timer->Elapsed();

		timer->Start();
		// Launch a kernel on the GPU 
		reduceKernel1 <<<blocks, maxThreads, maxThreads * sizeof(unsigned int) >>> (d_out, d_in, size);

		////reduceKernel <<<blocks, maxThreads, maxThreads * sizeof(unsigned int) >>> (d_intermediate, d_in, size);

		////while (blocks > maxThreads) {
		////	int maxSize = blocks;
		////	blocks = std::ceil((float)blocks / maxThreads);
		////	
		////	////checkCudaErrors(hipMalloc((void **)&d_out, blocks * sizeof(unsigned int)));
		////	////reduceKernel <<<blocks, maxThreads, maxThreads * sizeof(unsigned int) >>> (d_out, d_intermediate, maxSize);
		////	////checkCudaErrors(hipMemcpy(d_intermediate, d_out, blocks * sizeof(int), hipMemcpyDeviceToDevice));
		////	////hipFree(d_out);

		////	reduceKernel <<<blocks, maxThreads, maxThreads * sizeof(unsigned int) >>> (d_intermediate, d_intermediate, maxSize);
		////}

		////checkCudaErrors(hipMalloc((void **)&d_out, sizeof(unsigned int)));
		////reduceKernel <<<1, blocks, blocks * sizeof(unsigned int) >>> (d_out, d_intermediate, blocks);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		timer->Stop();
		kernelTime = timer->Elapsed();

		// Copy output vector from GPU buffer to host memory.
		checkCudaErrors(hipMemcpy(out, d_out, sizeof(unsigned int), hipMemcpyDeviceToHost));

		printf("(CUDA Timer) Parallel code delay: %f secs.\n", delay / 1000);
		printf("(CUDA Timer) Parallel code kernel: %f secs.\n", kernelTime / 1000);
	}
	catch (hipError_t ex)
	{
		cudaStatus = ex;
		goto FREE;
	};

FREE:
	hipFree(d_in);
	hipFree(d_out);
	////hipFree(d_intermediate);
	delete timer;

	return cudaStatus;
}

hipError_t reduceWithCuda(unsigned int *out, unsigned int **in, int nSegment, int sizeSegment, int size) {

	const int ARRAY_BYTES = sizeSegment * sizeof(unsigned int);
	unsigned int* d_in = 0;
	unsigned int* d_out = 0;
	unsigned int h_out = 0;
	hipError_t cudaStatus = hipSuccess;

	GpuTimer* timer = new GpuTimer();
	double delay = 0;
	double kernelTime = 0;

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		checkCudaErrorsAndExit(hipSetDevice(0));

		int maxThreads = sizeSegment;
		int blocks = 1;

		if (sizeSegment > MAX_THREADS_BLOCK) {
			maxThreads = MAX_THREADS_BLOCK;
			blocks = std::ceil((float)sizeSegment / maxThreads);
		}

	
		// Allocate GPU buffers for 2 vectors (one input, one output)    .
		checkCudaErrors(hipMalloc((void**)&d_in, ARRAY_BYTES));
		checkCudaErrors(hipMalloc((void**)&d_out, sizeof(unsigned int)));

		for (int i = 0; i < nSegment; i++) 
		{
			timer->Start();
			// Copy input vectors from host memory to GPU buffers.
			checkCudaErrors(hipMemcpy(d_in, in[i], ARRAY_BYTES, hipMemcpyHostToDevice));
			timer->Stop();
			delay += timer->Elapsed();

			int maxSize = sizeSegment;
			if (i == nSegment - 1)
			{	
				maxSize = size - sizeSegment * i;
			}
		
			timer->Start();
			// Launch a kernel on the GPU 
			reduceKernel1 <<<blocks, maxThreads, maxThreads * sizeof(unsigned int) >>> (d_out, d_in, maxSize);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			timer->Stop();
			kernelTime += timer->Elapsed();

			// Copy output vector from GPU buffer to host memory.
			checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(unsigned int), hipMemcpyDeviceToHost));
			std::cout << *out << std::endl;
			(*out) += h_out;
			checkCudaErrors(hipMemset(d_out, 0, sizeof(unsigned int)));
		}

		printf("(CUDA Timer) Parallel code delay: %f secs.\n", delay / 1000);
		printf("(CUDA Timer) Parallel code kernel: %f secs.\n", kernelTime / 1000);
	}
	catch (hipError_t ex)
	{
		cudaStatus = ex;
		goto FREE;
	};

FREE:
	hipFree(d_in);
	hipFree(d_out);
	delete timer;	

	return cudaStatus;
}


hipError_t reduceWithCudaEnd(unsigned int *out, unsigned int **in, int nSegment, int sizeSegment, int size) {

	const int ARRAY_BYTES = sizeSegment * sizeof(unsigned int);
	unsigned int* d_in = 0;
	unsigned int* d_out = 0;
	unsigned int* d_intermediate = 0;
	unsigned int h_out = 0;
	hipError_t cudaStatus = hipSuccess;

	GpuTimer* timer = new GpuTimer();
	double delay = 0;
	double kernelTime = 0;

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		checkCudaErrorsAndExit(hipSetDevice(0));

		int THREADS = sizeSegment;
		int BLOCKS = 1;

		if (sizeSegment > MAX_THREADS_BLOCK) {
			THREADS = MAX_THREADS_BLOCK;
			BLOCKS = std::ceil((float)sizeSegment / THREADS);
		}


		// Allocate GPU buffers for 2 vectors (one input, one output)    .
		checkCudaErrors(hipMalloc((void**)&d_in, ARRAY_BYTES));	
		checkCudaErrors(hipMalloc((void**)&d_out, sizeof(unsigned int)));
		checkCudaErrors(hipMalloc((void**)&d_intermediate, ARRAY_BYTES));

		for (int i = 0; i < nSegment; i++)
		{
			int blocks = BLOCKS;

			timer->Start();
			// Copy input vectors from host memory to GPU buffers.
			checkCudaErrors(hipMemcpy(d_in, in[i], ARRAY_BYTES, hipMemcpyHostToDevice));
			timer->Stop();
			delay += timer->Elapsed();

			int maxSize = sizeSegment;
			if (i == nSegment - 1)
			{
				maxSize = size - sizeSegment * i;
			}

			timer->Start();
			// Launch a kernel on the GPU 
			reduceKernel2 <<<blocks, THREADS, THREADS * sizeof(unsigned int) >>> (d_intermediate, d_in, maxSize);

			while (blocks > THREADS) {
				int maxSize = blocks;
				blocks = std::ceil((float)blocks / THREADS);
				reduceKernel2 <<<blocks, THREADS, THREADS * sizeof(unsigned int) >>> (d_intermediate, d_intermediate, maxSize);
			}

			checkCudaErrors(hipMalloc((void **)&d_out, sizeof(unsigned int)));
			reduceKernel2 <<<1, blocks, blocks * sizeof(unsigned int) >>> (d_out, d_intermediate, blocks);

			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			timer->Stop();
			kernelTime += timer->Elapsed();

			// Copy output vector from GPU buffer to host memory.
			checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(unsigned int), hipMemcpyDeviceToHost));
			(*out) += h_out;
			checkCudaErrors(hipMemset(d_out, 0, sizeof(unsigned int)));
			checkCudaErrors(hipMemset(d_intermediate, 0, ARRAY_BYTES));
		}

		printf("(CUDA Timer) Parallel code delay: %f secs.\n", delay / 1000);
		printf("(CUDA Timer) Parallel code kernel: %f secs.\n", kernelTime / 1000);
	}
	catch (hipError_t ex)
	{
		cudaStatus = ex;
		goto FREE;
	};

FREE:
	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_intermediate);
	delete timer;

	return cudaStatus;
}