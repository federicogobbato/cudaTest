#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_ptr.h"
#include "thrust/device_malloc.h"
#include "thrust/device_free.h"
#include "thrust/device_vector.h"
#include "thrust/sort.h"
#include "thrust/iterator/constant_iterator.h"
#include "thrust/extrema.h"
#include "utils.h"
#include "timer.h"

#include <stdio.h>
#include <algorithm>
#include <exception>

struct wordKeys {
	char k1, k2, k3, k4, k5, k6, k7, k8, k9;
};

__global__ void FindAllWords1(char **d_out, const char *d_in, int size, int sizeWord) {

	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < size)
	{
		char* newWord = (char*)malloc(sizeWord * sizeof(char)); 
		for (int i = 0; i < sizeWord; i++)
		{
			//d_out[id][i] = d_in[id + i];
			newWord[i] = d_in[id + i];
		}
		//d_out[id][sizeWord] = '\0';
		newWord[sizeWord] = '\0';
		d_out[id] = newWord;
	}
}

__global__ void CopyWords(char **d_in, char** d_out) {

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	printf("%s \n", d_in[id]);
	memcpy(d_out[id], d_in[id], 10);
}

__global__ void FreeWords(char **d_in, int size) {

	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < size)
	{
		free(d_in[id]);
	}
}

__global__ void FindAllWordsShared1(char **d_out, const char *d_in, int size, int sizeWord) {

	const int RADIUS = sizeWord;

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int tindex = threadIdx.x;

	extern __shared__ char temp[];
	temp[tindex] = d_in[id];
	if (threadIdx.x < RADIUS) {
		temp[tindex + blockDim.x] = d_in[id + blockDim.x];
	}
	__syncthreads();

	if (id < size)
	{
		//?d_out[id] = (char*)malloc(sizeWord * sizeof(char));
		for (int i = 0; i < sizeWord; i++)
		{			
			d_out[id][i] = temp[tindex + i];
		}	
		d_out[id][sizeWord] = '\0';
	}
}


__global__ void FindAllWordsShared2(char *d_out, const char *d_in, int size, int sizeWord) {

	const int RADIUS = sizeWord;

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int tindex = threadIdx.x;

	extern __shared__ char temp[];
	temp[tindex] = d_in[id];
	if (threadIdx.x < RADIUS) {
		temp[tindex + blockDim.x] = d_in[id + blockDim.x];
	}
	__syncthreads();

	if (id < size)
	{
		for (int i = 0; i < sizeWord; i++)
		{
			d_out[(id * sizeWord) + i] = temp[tindex + i];
		}
	}
}


////__global__ void FindAllWords2(wordKeys **d_out, const char *d_in, int size) {
////
////	int id = threadIdx.x + blockDim.x * blockIdx.x;
////
////	if (id < size) {
////		d_out[id]->k1 = d_in[id];
////		d_out[id]->k2 = d_in[id + 1];
////		d_out[id]->k3 = d_in[id + 2];
////		d_out[id]->k4 = d_in[id + 3];
////		d_out[id]->k5 = d_in[id + 4];
////		d_out[id]->k6 = d_in[id + 5];
////		d_out[id]->k7 = d_in[id + 6];
////		d_out[id]->k8 = d_in[id + 7];
////		d_out[id]->k9 = d_in[id + 8];
////	}
////}
////
////struct sort_wordKeys {
////	__host__ __device__ bool operator()(wordKeys &keys1, wordKeys &keys2) {
////		if (keys1.k1 < keys2.k1) return true;
////		if (keys1.k1 > keys2.k1) return false;
////		if (keys1.k2 < keys2.k2) return true;
////		if (keys1.k2 > keys2.k2) return false;
////		if (keys1.k3 < keys2.k3) return true;
////		if (keys1.k3 > keys2.k3) return false;
////		if (keys1.k4 < keys2.k4) return true;
////		if (keys1.k4 > keys2.k4) return false;
////		if (keys1.k5 < keys2.k5) return true;
////		if (keys1.k5 > keys2.k5) return false;
////		if (keys1.k6 < keys2.k6) return true;
////		if (keys1.k6 > keys2.k6) return false;
////		if (keys1.k7 < keys2.k7) return true;
////		if (keys1.k7 > keys2.k7) return false;
////		if (keys1.k8 < keys2.k8) return true;
////		if (keys1.k8 > keys2.k8) return false;
////		if (keys1.k9 < keys2.k9) return true;
////		return false;
////	}
////};
////
////struct equal_wordKeys {
////	__host__ __device__ bool operator()(wordKeys keys1, wordKeys keys2) {
////		if ((keys1.k1 == keys2.k1) && 
////			(keys1.k2 == keys2.k2) &&
////			(keys1.k3 == keys2.k3) &&
////			(keys1.k4 == keys2.k4) && 
////			(keys1.k5 == keys2.k5) &&
////			(keys1.k6 == keys2.k6) && 
////			(keys1.k7 == keys2.k7) &&
////			(keys1.k8 == keys2.k8) &&
////			(keys1.k9 == keys2.k9)) return true;
////		return false;
////	}
////};


struct charArrayCompare {
	__host__ __device__ bool operator()(const char* o1, const char* o2) {
		while ((*o1) && (*o1 == *o2))
		{
			o1++;
			o2++;
		}
		int diff = *(const unsigned char*)o1 - *(const unsigned char*)o2;
		return diff < 0;
	}
};

struct charArrayEqual {
	__host__ __device__ bool operator()(const char* o1, const char* o2) {
		while (*o1)
		{
			if (*o1 != *o2)
				return false;
			o1++;
			o2++;
		}
		return true;
	}
};


void FMFW1(const char* const in, const int& sizeWord, const unsigned int size) {

	clock_t tStart = clock();
	clock_t tDelay;
	int Delay = 0;

	const int N_WORDS = size - sizeWord + 1;
	const int ARRAY_BYTES = size * sizeof(char);
	const int WORD_BYTES = (sizeWord + 1) * sizeof(char);
	char* d_in = nullptr;
	char** d_listWords = nullptr;

	char** h_MostFrequentWords = nullptr;
	int N_MFWORDS = 0;

	checkCudaErrorsAndExit(hipSetDevice(0));

	try {

		int maxThreads = size;
		int blocks = 1;

		if (size > 1024) {
			maxThreads = 1024;
			blocks = std::ceil((float)size / maxThreads);
		}

		checkCudaErrors(hipMalloc((void **)&d_in, ARRAY_BYTES));
		checkCudaErrors(hipMemcpy(d_in, in, ARRAY_BYTES, hipMemcpyHostToDevice));

		tDelay = clock();

		checkCudaErrors(hipMalloc((void **)&d_listWords, N_WORDS * sizeof(char*)));
		////checkCudaErrors(hipHostAlloc((void **)&d_listWords, N_WORDS * sizeof(char*), hipMemoryTypeManaged));
		////for (int i = 0; i < N_WORDS; i++)
		////{
		////	checkCudaErrors(hipHostAlloc((void**)&(d_listWords[i]), WORD_BYTES, hipMemoryTypeManaged));
		////}

		Delay = clock() - tDelay;

		//FindAllWordsShared1 <<<blocks, maxThreads, (maxThreads + sizeWord) * sizeof(char)>>> (d_listWords, d_in, N_WORDS, sizeWord);
		FindAllWords1 << <blocks, maxThreads >>> (d_listWords, d_in, N_WORDS, sizeWord);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		//!Sort and reduce with Thrust.........

		//!Use a device_vector SLOW a lot the process, better use device_ptr
		thrust::device_ptr<char*> dev_ptr(d_listWords);
		thrust::sort(dev_ptr, dev_ptr + N_WORDS, charArrayCompare());

		thrust::device_vector<char*>d_Words(N_WORDS);
		thrust::device_vector<int>d_Frequency(N_WORDS);

		thrust::pair<thrust::device_vector<char*>::iterator, thrust::device_vector<int>::iterator> new_end;
		new_end = thrust::reduce_by_key(dev_ptr, dev_ptr + N_WORDS, thrust::constant_iterator<int>(1), d_Words.begin(), d_Frequency.begin(), charArrayEqual());

		int rsize = new_end.first - d_Words.begin();

		d_Words.erase(new_end.first, d_Words.end());
		d_Frequency.erase(new_end.second, d_Frequency.end());

		thrust::sort_by_key(d_Frequency.begin(), d_Frequency.end(), d_Words.begin(), thrust::greater<int>());

		thrust::device_vector<int>::iterator d_MostFrequent = thrust::max_element(d_Frequency.begin(), d_Frequency.end());

		//Print most frequent words
		thrust::device_ptr<char*> d_MostFrequentWords = thrust::device_malloc(rsize);
		N_MFWORDS = 0;
		for (int i = 0; i < rsize; i++) {
			if (d_Frequency[i] != *d_MostFrequent) {
				break;
			}
			d_MostFrequentWords[i] = d_Words[i];
			N_MFWORDS++;
		}

		checkCudaErrors(hipHostAlloc((void **)&h_MostFrequentWords, N_MFWORDS * sizeof(char*), hipMemoryTypeManaged));
		for (int i = 0; i < N_WORDS; i++)
		{
			checkCudaErrors(hipHostAlloc((void**)&(h_MostFrequentWords[i]), WORD_BYTES, hipMemoryTypeManaged));
		}

		CopyWords <<<1, N_MFWORDS >>> (thrust::raw_pointer_cast(d_MostFrequentWords), h_MostFrequentWords);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		////checkCudaErrors(hipMemcpy(h_MostFrequentWords, thrust::raw_pointer_cast(d_MostFrequentWords), N_MFWORDS * sizeof(char*), hipMemcpyDeviceToHost));
		////for (int i = 0; i < N_WORDS; i++)
		////{
		////	checkCudaErrors(hipMemcpy(h_MostFrequentWords[i], thrust::raw_pointer_cast(d_MostFrequentWords)[i], N_MFWORDS * sizeof(char*), hipMemcpyDeviceToHost));
		////}

		for (int i = 0; i < N_MFWORDS; i++) {
			printf("%s \n", h_MostFrequentWords[i]);
		}

		FreeWords <<<blocks, maxThreads >>> (d_listWords, N_WORDS);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}
	catch (hipError_t e)
	{
		hipFree(d_in);
		hipFree(d_listWords);
		for (int i = 0; i < N_MFWORDS; i++)
			hipHostFree(h_MostFrequentWords[i]);
		hipHostFree(h_MostFrequentWords);
		return;
	};

	hipFree(d_in);
	hipFree(d_listWords);

	printf("The parallel FMFW2 ran in %d ticks: %f secs.\n", clock() - tStart - Delay, ((double)(clock() - tStart - Delay)) / CLOCKS_PER_SEC);
	printf("Delay code is %d ticks: %f secs.\n\n", Delay, (double)Delay / CLOCKS_PER_SEC);
}

